#include "hip/hip_runtime.h"
#include <stdio.h>
#include "mmio.h"

// print vector values and device number
__global__ void print_vec_info(const int device_num, double* vec)
{
	int thrd_num = threadIdx.x;
	printf(" device: %d, vec[%d] = %lf\n", device_num, thrd_num, vec[thrd_num]);
}
// get the number of rows, cols and non-zero elements and set indices and values from Matrix Market file 
void read_mm(char* f_name, int* num_row, int* num_col, int* nnz, int* ind_row_coo, int* ind_col_coo, double* vals)
{
	FILE *fptr = fopen(f_name, "r");
	MM_typecode matcode;
	mm_read_banner(fptr, &matcode);
	if(mm_is_complex(matcode) && mm_is_matrix(matcode) && mm_is_sparse(matcode)){
		printf("Matrix Market type: [%s]\n", mm_typecode_to_str(matcode));
		exit(1);
	}
	mm_read_mtx_crd_size(fptr, num_row, num_col, nnz);
	size_t size = (*nnz) * sizeof(int);
	ind_row_coo = (int*)malloc(size);
	ind_col_coo = (int*)malloc(size);
	size = (*nnz) * sizeof(double);
	vals = (double*)malloc(size);
	for(int i = 0; i < (*nnz); ++i){
		fscanf(fptr, "%d %d %lf\n", &ind_row_coo[i], &ind_col_coo[i], &vals[i]);
		ind_row_coo[i]--;
		ind_col_coo[i]--;
	}
	fclose(fptr);
}

int main(int argc, char* argv[])
{
	if(argc < 2){
		fprintf(stderr, "Usage: %s [matrix-market-filename]\n", argv[0]);
		exit(1);
	}

	// process Matrix Market file and set some values
	int num_row, num_col, nnz;
	int *ind_row_coo=NULL, *ind_col_coo=NULL;
	double* vals=NULL;
	read_mm(argv[1], &num_row, &num_col, &nnz, ind_row_coo, ind_col_coo, vals);

	// prepare vector-x and -y for y = Ax
	size_t size = num_col * sizeof(double);
	double* vec_x = (double*)malloc(size);
	for(int i = 0; i < num_col; ++i){
		vec_x[i] = 1.0;
	}
	printf("here\n");
	size = num_row * sizeof(double);
	double* vec_y = (double*)malloc(size);
	for(int i = 0; i < num_row; ++i){
		vec_y[i] = 0.0;
	}

	int num_device;
	hipGetDeviceCount(&num_device);
	printf("num_device = %d\n", num_device);
	// allocate device memory for vec-x and -y
	//double *vec_x_d=NULL, *vec_y_d=NULL;
	//size = num_col * sizeof(double);
	//hipMalloc(vec_x_d, size);
	double **vec_x_ds = (double**)malloc(num_device*sizeof(double*));
	double **vec_y_ds = (double**)malloc(num_device*sizeof(double*));
	for(int i = 0; i < num_device; ++i){
		hipSetDevice(i);
		size_t size_x = num_col * sizeof(double);
		hipMalloc(&vec_x_ds[i], size_x);
		hipMemcpy(vec_x_ds[i], vec_x, size_x, hipMemcpyHostToDevice);
		size_t size_y = num_row * sizeof(double);
		hipMalloc(&vec_y_ds[i], size_y);
		hipMemcpy(vec_y_ds[i], vec_y, size_y, hipMemcpyHostToDevice);
	}

	// on each device print the vector and device number
	dim3 dimGrid(1, 1);
	dim3 dimBlock(num_col, 1);
	for(int i = 0; i < num_device; ++i){
		print_vec_info<<<dimGrid, dimBlock>>>(i, vec_x_ds[i]);
	}


	// free pointers allocated on each device
	for(int i = 0; i < num_device; ++i){
		hipFree(vec_x_ds[i]);
		hipFree(vec_y_ds[i]);
	}
	free(vec_y_ds);
	free(vec_x_ds);
	free(vec_y);
	free(vec_x);
	return 0;
}
